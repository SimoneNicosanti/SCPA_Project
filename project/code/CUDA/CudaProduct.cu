// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include "Matrix.h"
#include "CudaProduct.h"
#include "Kernel_4.cuh"

#define DEF_MB 50
#define DEF_NB 50

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrix, int rows, int cols, size_t *pitchPtr) ;
float callKernel(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC) ;


float callKernel(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC) {

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    // Implem_4
    const int M_BLOCK_SIZE = 128 ;
    const int N_BLOCK_SIZE = 128 ;
    const int K_BLOCK_SIZE = 8 ;

    const int A_TILE_SIZE = 4 ;
    const int B_TILE_SIZE = 4 ;

    dim3 BLOCK_DIM((M_BLOCK_SIZE * N_BLOCK_SIZE) / (A_TILE_SIZE * B_TILE_SIZE)) ;
    dim3 GRID_DIM(((n - 1) / N_BLOCK_SIZE) + 1, ((m - 1) / M_BLOCK_SIZE) + 1) ;

    timer->start();
    gpuProduct_4
    <M_BLOCK_SIZE, K_BLOCK_SIZE, N_BLOCK_SIZE, A_TILE_SIZE, B_TILE_SIZE> 
    <<<GRID_DIM, BLOCK_DIM>>>(
        A, B, C, 
        m, k, n, 
        pitchA, pitchB, pitchC
    );
    checkCudaErrors(hipDeviceSynchronize());
    timer->stop();

    return timer->getTime() ;

}

void CudaProduct(
    Matrix hostA, Matrix hostB, Matrix hostC, 
    int m, int k, int n, 
    int mb, int nb, 
    Info *infoPtr
) {

    if (mb <= 0) {
        mb = DEF_MB ;
    }
    if (nb <= 0) {
        nb = DEF_NB ;
    }

    Matrix devA, devB, devC ;
    size_t pitchA, pitchB, pitchC ;
    moveMatricesFromHostToDevice(hostA, &devA, m, k, &pitchA) ;
    moveMatricesFromHostToDevice(hostB, &devB, k, n, &pitchB) ;
    moveMatricesFromHostToDevice(hostC, &devC, m, n, &pitchC) ;

    float kernelTime = callKernel(devA, devB, devC, m, k, n, pitchA, pitchB, pitchC) ;
    infoPtr->productTime = kernelTime ;

    checkCudaErrors(
        hipMemcpy2D(hostC, sizeof(MatrixElemType) * n, devC, pitchC * sizeof(MatrixElemType), sizeof(MatrixElemType) * n, m, hipMemcpyDeviceToHost)
    ) ;
    // checkCudaErrors(
    //     hipMemcpy(hostC, devC, sizeof(MatrixElemType) * m * n, hipMemcpyDeviceToHost) 
    // ) ;

    hipFree(devA) ;
    hipFree(devB) ;
    hipFree(devC) ;

    return ;
}

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrixPtr, int rows, int cols, size_t *pitchPtr) {
    checkCudaErrors(
        hipHostRegister(hostMatrix, sizeof(MatrixElemType) * rows * cols, hipHostRegisterDefault)
    ) ;

    // checkCudaErrors(
    //     hipMalloc((void **) devMatrixPtr, sizeof(MatrixElemType) * rows * cols)
    // ) ;
    // checkCudaErrors(
    //     hipMemcpy(*devMatrixPtr, hostMatrix, sizeof(MatrixElemType) * rows * cols, hipMemcpyHostToDevice) 
    // ) ;
    //*pitchPtr = 0 ;

    checkCudaErrors(
        hipMallocPitch((void **) devMatrixPtr, pitchPtr, sizeof(MatrixElemType) * cols, rows)
    ) ;
    checkCudaErrors(
        hipMemcpy2D(*devMatrixPtr, *pitchPtr, hostMatrix, sizeof(MatrixElemType) * cols, sizeof(MatrixElemType) * cols, rows, hipMemcpyHostToDevice)
    ) ;
    *pitchPtr = *pitchPtr / sizeof(MatrixElemType) ;
}