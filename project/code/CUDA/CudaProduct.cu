#include "hip/hip_runtime.h"
// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "Matrix.h"
#include "CudaProduct.h"

#define DEF_MB 50
#define DEF_NB 50

const dim3 BLOCK_DIM(16, 8) ;

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrix, int rows, int cols, size_t *pitchPtr) ;


__global__ void gpuProduct(Matrix A, Matrix B, Matrix C, int m, int k , int n, int pitchA, int pitchB, int pitchC) {

    __shared__ MatrixElemType subA[BLOCK_DIM.y][32] ;
    __shared__ MatrixElemType subB[32][BLOCK_DIM.x] ;

    float subCElem = 0.0 ;
    
    int rowA = threadIdx.y + blockIdx.y * blockDim.y ;
    int colB = threadIdx.x + blockIdx.x * blockDim.x ;

    if (rowA < m && colB < n) {
        for (int kMult = 0 ; kMult < (k / 32) + 1 ; kMult++) {

            for (int kLocIdx = 0 ; kLocIdx < min(k - 32 * kMult, 32) ; kLocIdx++) {
                int kGlobIdx = kMult * 32 + kLocIdx ;
                subA[threadIdx.y][kLocIdx] = A[INDEX(rowA, kGlobIdx , pitchA)] ;
                subB[kLocIdx][threadIdx.x] = B[INDEX(kGlobIdx, colB, pitchB)] ;
            }

            for (int i = 0 ; i < min(k - 32 * kMult, 32) ; i++) {
                subCElem += subA[threadIdx.y][i] * subB[i][threadIdx.x] ;
            }
        }
        C[INDEX(rowA, colB, pitchC)] += subCElem ;
        
    }
    
}


void CudaProduct(Matrix hostA, Matrix hostB, Matrix hostC, int m, int k, int n, int mb, int nb, Info *infoPtr) {

    if (mb <= 0) {
        mb = DEF_MB ;
    }
    if (nb <= 0) {
        nb = DEF_NB ;
    }

    Matrix devA, devB, devC ;
    size_t pitchA, pitchB, pitchC ;
    moveMatricesFromHostToDevice(hostA, &devA, m, k, &pitchA) ;
    moveMatricesFromHostToDevice(hostB, &devB, k, n, &pitchB) ;
    moveMatricesFromHostToDevice(hostC, &devC, m, n, &pitchC) ;

    dim3 GRID_DIM(((n - 1) / BLOCK_DIM.x) + 1, ((m - 1) / BLOCK_DIM.y) + 1) ;

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    gpuProduct<<<GRID_DIM, BLOCK_DIM>>>(devA, devB, devC, m, k, n, pitchA, pitchB, pitchC);
    checkCudaErrors(hipDeviceSynchronize());
    timer->stop();
    
    infoPtr->productTime = timer->getTime() ;

    checkCudaErrors(
        hipMemcpy2D(hostC, sizeof(MatrixElemType) * m, devC, pitchC * sizeof(MatrixElemType), sizeof(MatrixElemType) * m, n, hipMemcpyDeviceToHost)
    ) ;
    // checkCudaErrors(
    //     hipMemcpy(hostC, devC, sizeof(MatrixElemType) * m * n, hipMemcpyDeviceToHost) 
    // ) ;

    hipFree(devA) ;
    hipFree(devB) ;
    hipFree(devC) ;

    return ;
}

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrixPtr, int rows, int cols, size_t *pitchPtr) {
    checkCudaErrors(
        hipHostRegister(hostMatrix, sizeof(MatrixElemType) * rows * cols, hipHostRegisterDefault)
    ) ;
    checkCudaErrors(
        hipMallocPitch((void **) devMatrixPtr, pitchPtr, sizeof(MatrixElemType) * cols, rows)
    ) ;
    // checkCudaErrors(
    //     hipMalloc((void **) devMatrixPtr, sizeof(MatrixElemType) * rows * cols)
    // ) ;

    checkCudaErrors(
        hipMemcpy2D(*devMatrixPtr, *pitchPtr, hostMatrix, sizeof(MatrixElemType) * cols, sizeof(MatrixElemType) * cols, rows, hipMemcpyHostToDevice)
    ) ;
    // checkCudaErrors(
    //     hipMemcpy(*devMatrixPtr, hostMatrix, sizeof(MatrixElemType) * rows * cols, hipMemcpyHostToDevice) 
    // ) ;
    *pitchPtr = *pitchPtr / sizeof(MatrixElemType) ;
    //*pitchPtr = 0 ;
}