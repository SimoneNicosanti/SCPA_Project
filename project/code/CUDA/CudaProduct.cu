// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include "Matrix.h"
#include "CudaProduct.h"

#include "Kernel_4.cuh"
#include "Kernel_5.cuh"
#include "Kernel_6.cuh"

#define DEF_MB 50
#define DEF_NB 50

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrix, int rows, int cols, size_t *pitchPtr) ;
float callKernel(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC, Version version) ;

void callKernel_4(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC) {
    printf("CUDA Product Version >>> 4\n") ;
    const int M_BLOCK_SIZE = 128 ;
    const int N_BLOCK_SIZE = 128 ;
    const int K_BLOCK_SIZE = 8 ;

    const int A_TILE_SIZE = 4 ;
    const int B_TILE_SIZE = 4 ;

    dim3 BLOCK_DIM((M_BLOCK_SIZE * N_BLOCK_SIZE) / (A_TILE_SIZE * B_TILE_SIZE)) ;
    dim3 GRID_DIM(((n - 1) / N_BLOCK_SIZE) + 1, ((m - 1) / M_BLOCK_SIZE) + 1) ;

    gpuProduct_4
        <M_BLOCK_SIZE, K_BLOCK_SIZE, N_BLOCK_SIZE, A_TILE_SIZE, B_TILE_SIZE> 
        <<<GRID_DIM, BLOCK_DIM>>>(
            A, B, C, 
            m, k, n, 
            pitchA, pitchB, pitchC
        );
    checkCudaErrors(hipDeviceSynchronize());
}

void callKernel_5(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC) {
    printf("CUDA Product Version >>> 5\n") ;
    const int M_BLOCK_SIZE = 128 ;
    const int N_BLOCK_SIZE = 128 ;
    const int K_BLOCK_SIZE = 16 ;

    const int A_TILE_SIZE = 8 ;
    const int B_TILE_SIZE = 8 ;

    dim3 BLOCK_DIM((M_BLOCK_SIZE * N_BLOCK_SIZE) / (A_TILE_SIZE * B_TILE_SIZE)) ;
    dim3 GRID_DIM(((n - 1) / N_BLOCK_SIZE) + 1, ((m - 1) / M_BLOCK_SIZE) + 1) ;

    gpuProduct_5
        <M_BLOCK_SIZE, K_BLOCK_SIZE, N_BLOCK_SIZE, A_TILE_SIZE, B_TILE_SIZE> 
        <<<GRID_DIM, BLOCK_DIM>>>(
            A, B, C, 
            m, k, n, 
            pitchA, pitchB, pitchC
        );
    checkCudaErrors(hipDeviceSynchronize());
}

void callKernel_6(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC) {
    // TODO Implementare Warp Tiling ???
    printf("CUDA Product Version >>> 5\n") ;
    const int M_BLOCK_SIZE = 128 ;
    const int N_BLOCK_SIZE = 128 ;
    const int K_BLOCK_SIZE = 16 ;

    const int A_TILE_SIZE = 8 ;
    const int B_TILE_SIZE = 8 ;

    dim3 BLOCK_DIM((M_BLOCK_SIZE * N_BLOCK_SIZE) / (A_TILE_SIZE * B_TILE_SIZE)) ;
    dim3 GRID_DIM(((n - 1) / N_BLOCK_SIZE) + 1, ((m - 1) / M_BLOCK_SIZE) + 1) ;

    gpuProduct_6
        <M_BLOCK_SIZE, K_BLOCK_SIZE, N_BLOCK_SIZE, A_TILE_SIZE, B_TILE_SIZE> 
        <<<GRID_DIM, BLOCK_DIM>>>(
            A, B, C, 
            m, k, n, 
            pitchA, pitchB, pitchC
        );
    checkCudaErrors(hipDeviceSynchronize());
}

float callKernel(Matrix A, Matrix B, Matrix C, int m, int k, int n, int pitchA, int pitchB, int pitchC, Version version) {

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    switch (version)
    {
    case FOUR:
        callKernel_4(A, B, C, m, k, n, pitchA, pitchB, pitchC) ;
        break;
    case FIVE:
        callKernel_5(A, B, C, m, k, n, pitchA, pitchB, pitchC) ;
        break;
    case SIX:
        callKernel_6(A, B, C, m, k, n, pitchA, pitchB, pitchC) ;
        break;
    case DEFAULT:
        callKernel_6(A, B, C, m, k, n, pitchA, pitchB, pitchC) ;
    }
    timer->stop();    

    return timer->getTime() ;

}

void CudaProduct(
    Matrix hostA, Matrix hostB, Matrix hostC, 
    int m, int k, int n, 
    int mb, int nb, 
    Version version,
    Info *infoPtr
) {

    if (mb <= 0) {
        mb = DEF_MB ;
    }
    if (nb <= 0) {
        nb = DEF_NB ;
    }

    Matrix devA, devB, devC ;
    size_t pitchA, pitchB, pitchC ;
    moveMatricesFromHostToDevice(hostA, &devA, m, k, &pitchA) ;
    moveMatricesFromHostToDevice(hostB, &devB, k, n, &pitchB) ;
    moveMatricesFromHostToDevice(hostC, &devC, m, n, &pitchC) ;

    float kernelTime = callKernel(devA, devB, devC, m, k, n, pitchA, pitchB, pitchC, version) ;
    infoPtr->productTime = kernelTime ;

    checkCudaErrors(
        hipMemcpy2D(hostC, sizeof(MatrixElemType) * n, devC, pitchC * sizeof(MatrixElemType), sizeof(MatrixElemType) * n, m, hipMemcpyDeviceToHost)
    ) ;

    hipFree(devA) ;
    hipFree(devB) ;
    hipFree(devC) ;

    return ;
}

void moveMatricesFromHostToDevice(Matrix hostMatrix, Matrix *devMatrixPtr, int rows, int cols, size_t *pitchPtr) {
    checkCudaErrors(
        hipHostRegister(hostMatrix, sizeof(MatrixElemType) * rows * cols, hipHostRegisterDefault)
    ) ;
    checkCudaErrors(
        hipMallocPitch((void **) devMatrixPtr, pitchPtr, sizeof(MatrixElemType) * cols, rows)
    ) ;
    checkCudaErrors(
        hipMemcpy2D(*devMatrixPtr, *pitchPtr, hostMatrix, sizeof(MatrixElemType) * cols, sizeof(MatrixElemType) * cols, rows, hipMemcpyHostToDevice)
    ) ;
    *pitchPtr = *pitchPtr / sizeof(MatrixElemType) ;
}